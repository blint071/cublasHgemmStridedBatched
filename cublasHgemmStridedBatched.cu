
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "math.h"
#include "hipblas.h" 
#include <hip/hip_fp16.h>
#include <iostream>
#include <sys/time.h>


//nvcc -lcublas cublas.c -o cublas.out

void main2()
{
int i,j,k,index;

// Linear dimension of matrices
int dim = 100;
int dim2 = 1;
int batch_count = 10000;
 
// Allocate host storage for batch_count A,B,C square matrices
half* h_A = (half*)malloc(sizeof(half) * dim2 * dim * batch_count);
half* h_B = (half*)malloc(sizeof(half) * dim * dim * batch_count);
half* h_C = (half*)malloc(sizeof(half) * dim * dim * batch_count);
    for(k=0; k<batch_count; k++) {
        for(j=0; j<dim; j++) {
                for(i=0; i<dim; i++) {
                index = i*dim + j + k*dim*dim;
                  //h_A[index] = index*index + 0.0f;
                  h_B[index] = index + 1.0f;
                  h_C[index] = 0.0f;
        }
    }
}

    for(k=0; k<batch_count; k++) {
        for(j=0; j<dim2; j++) {
                for(i=0; i<dim; i++) {
                index = i*dim + j + k*dim*dim;
                  h_A[index] = index*index + 0.0f;
        }
    }
}
 



half *d_A, *d_B, *d_C;
 
hipMalloc(&d_A, sizeof(half) * dim2 * dim * batch_count);
hipMalloc(&d_B, sizeof(half) * dim * dim * batch_count);
hipMalloc(&d_C, sizeof(half) * dim * dim * batch_count);
 
hipMemcpy(h_A,d_A,sizeof(half) * dim2 * dim * batch_count,hipMemcpyDeviceToHost);
hipMemcpy(h_B,d_B,sizeof(half) * dim * dim * batch_count,hipMemcpyDeviceToHost);
hipMemcpy(h_C,d_C,sizeof(half) * dim * dim * batch_count,hipMemcpyDeviceToHost);

hipblasHandle_t handle;
hipblasCreate(&handle);
printf("hi");  
// Do the actual multiplication 
 
struct timeval t1, t2;
half alpha = 1.0f;  half beta = 1.0f;
for (int za=0 ; za<50000; za++)
{  
    
hipblasHgemmStridedBatched(handle,
                              HIPBLAS_OP_N, 
                              HIPBLAS_OP_N,
                              dim, dim2, dim,
                              &alpha,
                              (const half*)d_A, dim,
                              dim2*dim,
                              (const half*)d_B, dim,
                              dim*dim,
                              &beta,
                              d_C, dim, 
                              dim*dim, 
                              batch_count);
}
 

hipMemcpy(h_C,d_C,sizeof(half) * dim * dim * batch_count,hipMemcpyDeviceToHost);
// Destroy the handle
hipblasDestroy(handle);


hipFree(d_A);
hipFree(d_B);
hipFree(d_C);
free(h_A);
free(h_B);
free(h_C);
}

int main(){
  
  main2();
  printf("Success!\n");
  return 0;
}